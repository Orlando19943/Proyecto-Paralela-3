#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <unistd.h>
#include "common/pgm.h"

const int degree_increment = 2;
const int total_degree_bins = 180 / degree_increment;
const int total_radial_bins = 100;
const int threads_per_block = 256;
const float degree_bin_width = degree_increment * M_PI / 180;

const int total_bins = total_degree_bins * total_radial_bins;

const char *BOLD = "\033[1m";
const char *RED = "\033[91m";
const char *CLEAR = "\033[0m";
const char *GREEN = "\033[92m";

#define START_GPU_TIMING() hipEventRecord(start)
#define END_GPU_TIMING(time)      \
    hipEventRecord(stop);        \
    hipEventSynchronize(stop);   \
    hipEventElapsedTime(time, start, stop)


void compare_results(int *cpu_results, int *in_device_results) {
    int *gpu_results = (int *) malloc(total_bins * sizeof(int));
    hipMemcpy(gpu_results, in_device_results, sizeof(int) * total_bins, hipMemcpyDeviceToHost);

    int i;
    int mismatch = 0;
    int rounding_erros = 0;
    for (i = 0; i < total_bins; i++) {
        if (cpu_results[i] != gpu_results[i]) {
            if (gpu_results[i] - 1 == cpu_results[i] || gpu_results[i] + 1 == cpu_results[i]) {
                rounding_erros += 1;
            } else {
                printf(" - Calculation mismatch at : %i %i %i\n", i, cpu_results[i], gpu_results[i]);
                mismatch++;
            }
        }
    }

    free(gpu_results);
    printf("Total possible rounding errors (±1): %i\n", rounding_erros);
    printf("Total mismatches: %i\n", mismatch);
}

void CPU_HoughTran(const unsigned char *picture, int width, int height, int **accumulator) {
    float image_diagonal_length = sqrt(1.0 * width * width + 1.0 * height * height) / 2;
    int x, y;
    float theta;

    *accumulator = new int[total_bins];
    memset(*accumulator, 0, sizeof(int) * total_bins);

    int x_center = width / 2;
    int y_center = height / 2;
    float radial_bin_width = 2 * image_diagonal_length / total_radial_bins;

    // for each pixel
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            int pixel = j * width + i;
            if (picture[pixel] > 0) {
                // votar por todas las líneas que pasan por ese punto
                x = i - x_center;   // medida desde el centro
                y = y_center - j;   // medida desde el centro invertida
                theta = 0;
                for (int degree_bin = 0; degree_bin < total_degree_bins; degree_bin++) {
                    float radius = x * cos(theta) + y * sin(theta);
                    int radial_bin = (radius + image_diagonal_length) / radial_bin_width;
                    int bin = radial_bin * total_degree_bins + degree_bin;
                    (*accumulator)[bin]++; //+1 para este radio radius y este theta
                    theta += degree_bin_width;
                }
            }
        }
    }
}

//*****************************************************************
__device__ __constant__ float pre_cosine[total_degree_bins];
__device__ __constant__ float pre_sin[total_degree_bins];

void precalculate_trigonometry(float **device_cosine, float **device_sin) {
    float *precomputed_cos = (float *) malloc(sizeof(float) * total_degree_bins);
    float *precomputed_sin = (float *) malloc(sizeof(float) * total_degree_bins);

    int i;
    float degree = 0;
    for (i = 0; i < total_degree_bins; i++) {
        precomputed_cos[i] = cos(degree);
        precomputed_sin[i] = sin(degree);

        degree += degree_bin_width;
    }

    // fill global const mem
    hipMemcpyToSymbol(HIP_SYMBOL(pre_cosine), precomputed_cos, sizeof(float) * total_degree_bins);
    hipMemcpyToSymbol(HIP_SYMBOL(pre_sin), precomputed_sin, sizeof(float) * total_degree_bins);

    // move to device
    hipMalloc((void **) device_cosine, sizeof(float) * total_degree_bins);
    hipMalloc((void **) device_sin, sizeof(float) * total_degree_bins);

    hipMemcpy(*device_cosine, precomputed_cos, sizeof(float) * total_degree_bins, hipMemcpyHostToDevice);
    hipMemcpy(*device_sin, precomputed_sin, sizeof(float) * total_degree_bins, hipMemcpyHostToDevice);

    free(precomputed_cos);
    free(precomputed_sin);
}


//*****************************************************************
__global__ void GPU_HoughTranShared(
        unsigned char *picture,
        int width, int height,
        int *accumulator,
        float image_diagonal_length,
        float radial_bin_width
) {
    __shared__ int partial_accumulator[total_bins];
    int local_id = threadIdx.x;
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    // set the bins the thread is responsible for initializing and copying to master
    int bins_per_thread = total_bins / threads_per_block;
    int bins_start = local_id * bins_per_thread;
    int bins_end = (local_id + 1) * bins_per_thread;
    if (local_id == threads_per_block - 1) {
        bins_end = total_degree_bins * total_radial_bins;
    }

    // initialize memory to zero
    for (int i = bins_start; i < bins_end; i++) {
        partial_accumulator[i] = 0;
    }

    __syncthreads();


    if (global_id >= width * height)
        return;

    int x_center = width / 2;
    int y_center = height / 2;

    int x = global_id % width - x_center;
    int y = y_center - global_id / width;

    // fill local poll table
    if (picture[global_id] > 0) {
        for (int bin_degree = 0; bin_degree < total_degree_bins; bin_degree++) {
            float radius = x * pre_cosine[bin_degree] + y * pre_sin[bin_degree];
            int radial_bin = (radius + image_diagonal_length) / radial_bin_width;
            atomicAdd_block(partial_accumulator + (radial_bin * total_degree_bins + bin_degree), 1);
        }
    }

    __syncthreads();

    // update global poll table
    for (int i = bins_start; i < bins_end; i++) {
        if (partial_accumulator[i] > 0) {
            atomicAdd(accumulator + i, partial_accumulator[i]);
        }
    }
}

__global__ void GPU_HoughTranConst(
        unsigned char *picture,
        int width,
        int height,
        int *accumulator,
        float image_diagonal_length,
        float radial_bin_width
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_id >= width * height)
        return;

    int x_center = width / 2;
    int y_center = height / 2;

    int x = global_id % width - x_center;
    int y = y_center - global_id / width;

    if (picture[global_id] > 0) {
        for (int bin_degree = 0; bin_degree < total_degree_bins; bin_degree++) {
            float radius = x * pre_cosine[bin_degree] + y * pre_sin[bin_degree];
            int radial_bin = (radius + image_diagonal_length) / radial_bin_width;
            atomicAdd(accumulator + (radial_bin * total_degree_bins + bin_degree), 1);
        }
    }
}

__global__ void GPU_HoughTran(
        unsigned char *picture,
        int width,
        int height,
        int *accumulator,
        float image_diagonal_length,
        float radial_bin_width,
        float *precomputed_cos,
        float *precomputed_sin
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_id >= width * height)
        return;

    int x_center = width / 2;
    int y_center = height / 2;

    int x = global_id % width - x_center;
    int y = y_center - global_id / width;

    if (picture[global_id] > 0) {
        for (int bin_degree = 0; bin_degree < total_degree_bins; bin_degree++) {
            // float radius = x * cos(bin_degree) + y * sin(bin_degree); //probar con esto para ver diferencia en tiempo
            float radius = x * precomputed_cos[bin_degree] + y * precomputed_sin[bin_degree];
            int radial_bin = (radius + image_diagonal_length) / radial_bin_width;
            atomicAdd(accumulator + (radial_bin * total_degree_bins + bin_degree), 1);
        }
    }
}

//*****************************************************************
int main(int argc, char **argv) {
    // get input from user
    if (argc < 2) {
        printf("Usage: %s image\n", argv[0]);
        exit(1);
    }
    printf("Loading image %s\n", argv[1]);

    // write image to mem
    PGMImage inImg(argv[1]);
    int width = inImg.x_dim;
    int height = inImg.y_dim;

    printf("Image size is %d x %d\n", width, height);

    // write the image to device
    unsigned char *image_in_device;
    hipMalloc((void **) &image_in_device, sizeof(unsigned char) * width * height);
    hipMemcpy(image_in_device, inImg.pixels, sizeof(unsigned char) * width * height, hipMemcpyHostToDevice);

    // pre-compute values to be stored
    int blockNum = ceil(width * height / threads_per_block);

    float *precomputed_cos, *precomputed_sin;
    precalculate_trigonometry(&precomputed_cos, &precomputed_sin);

    float max_radius = sqrt(1.0 * width * width + 1.0 * height * height) / 2;
    float radial_bin_width = 2 * max_radius / total_radial_bins;

    //  =======================================================================================
    //  ============================ START ACTUAL CALCULATIONS ================================
    //  =======================================================================================
    int *device_accumulator;
    hipMalloc((void **) &device_accumulator, sizeof(int) * total_degree_bins * total_radial_bins);

    // for time measuring
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // CPU calculation
    int *cpu_accumulator;
    CPU_HoughTran(inImg.pixels, width, height, &cpu_accumulator);

    // ------------------ GPU - No Shared Memory Nor constant memory -----------------
    printf("\n%s%sGPU - No const nor shared mem%s\n", BOLD, RED, CLEAR);
    hipMemset(device_accumulator, 0, sizeof(int) * total_bins);

    START_GPU_TIMING();
    GPU_HoughTran<<<blockNum, threads_per_block>>>(
            image_in_device,
            width,
            height,
            device_accumulator,
            max_radius,
            radial_bin_width,
            precomputed_cos,
            precomputed_sin
    );
    END_GPU_TIMING(&milliseconds);

    compare_results(cpu_accumulator, device_accumulator);
    printf("%sGPU time: %f ms%s\n", GREEN, milliseconds, CLEAR);

    // ------------------------------ GPU - const memory ------------------------------
    printf("\n%s%sGPU - Precomputed Sin and Cos (const memory) %s\n", BOLD, RED, CLEAR);
    hipMemset(device_accumulator, 0, sizeof(int) * total_bins);

    START_GPU_TIMING();
    GPU_HoughTranConst<<<blockNum, threads_per_block>>>(
            image_in_device,
            width,
            height,
            device_accumulator,
            max_radius,
            radial_bin_width
    );
    END_GPU_TIMING(&milliseconds);

    compare_results(cpu_accumulator, device_accumulator);
    printf("%sGPU time: %f ms%s\n", GREEN, milliseconds, CLEAR);

    // --------------------- GPU - const memory and shared memory --------------------
    printf("\n%s%sGPU - Shared memory %s\n", BOLD, RED, CLEAR);
    hipMemset(device_accumulator, 0, sizeof(int) * total_bins);

    START_GPU_TIMING();
    GPU_HoughTranShared<<<blockNum, threads_per_block>>>(
            image_in_device,
            width,
            height,
            device_accumulator,
            max_radius,
            radial_bin_width
    );
    END_GPU_TIMING(&milliseconds);

    compare_results(cpu_accumulator, device_accumulator);
    printf("%sGPU time: %f ms%s\n", GREEN, milliseconds, CLEAR);

    //  =======================================================================================
    //  ================================= END OF CALCULATIONS ================================
    //  =======================================================================================

    // save results
    inImg.to_jpg_with_line(
            "out/test.jpg",
            cpu_accumulator,
            4600,
            total_degree_bins,
            degree_increment,
            total_radial_bins
    );

    // cleanup
    free(cpu_accumulator);
    hipFree(image_in_device);
    hipFree(device_accumulator);
    hipFree(precomputed_sin);
    hipFree(precomputed_cos);

    return 0;
}
